#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "solver.h"
#include "cuda_solver.h"


void checkCuda(hipError_t result) 
{
	if (result != hipSuccess) {
	   printf("CUDA call failed.\n");
	   exit(1);
	}
 }

__global__ void lin_solve_kernel(int N, int b, fluid *x, fluid *x0, float a, float c) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	// tmp = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)]));
	// x[IX(i, j)] = tmp / c;
}

void lin_solve_cuda(int N, int b, fluid *x, fluid *x0, float a, float c, GPUSTATE gpu)
{
	
	int i, j, k;
	fluid tmp = 0;
	
	int threadBlockSize = 512;
	checkCuda(hipGetLastError());

	for (k = 0; k < 20; k++)
	{
		FOR_EACH_CELL
		tmp = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)]));
		x[IX(i, j)] = tmp / c;
		END_FOR
		set_bnd(N, b, x);
	}
}